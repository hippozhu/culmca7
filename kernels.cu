#include "hip/hip_runtime.h"
#include "mycommon.h"

#define BSIZE 256

__constant__ int nfeat;
__constant__ int ntrain;
__constant__ int ntest;
__constant__ int nvalidate;
__constant__ int nclass;
__constant__ int nnegibor;
__constant__ double d_mu;
__constant__ double d_nu[4];
__constant__ int idx_o;

__constant__ int *target;
__constant__ double *km_train;
__constant__ double *km_test;
__constant__ double *km_validate;
__constant__ double *O[2];
__constant__ double *t_target;
__constant__ double *t_triplet;
__constant__ double *t_update;

__constant__ short *label_train, *label_test, *label_validate;
__constant__ struct Inst *type_inst[4];
__constant__ unsigned typecount[4];
__constant__ int *target_offset;
__constant__ int nn[4];

__constant__ double *dist_target;
__constant__ double *dist1;
__constant__ double *dist2;
__constant__ double *hinge_val;

__constant__ double *dist_knn;
__constant__ int *ino_knn;
__constant__ int *neighbor_knn;

__device__ double f_val;
__device__ double sub_fval[84];
__device__ double acc_knn;
__device__ int hits[4];

__device__ void kernelMatrix(double *km, double *d1, int n1, double *d2, int n2){
  int ub = n1 * n2;
  int stride = blockDim.x * gridDim.x;
  double c_val;
  int i, j;
  for (int m = blockIdx.x * blockDim.x + threadIdx.x; m < ub; m += stride){
    i = m / n2;
	j = m % n2;
	c_val = .0;
	for (int n = 0; n < nfeat; ++ n)
	  c_val += pow(d1[n * n1 + i] - d2[n * n2 + j], 2);
	km[m] = exp(-c_val / nfeat);
  }
}

__global__ void calcKM(double *train, double *test, double *validate){
  kernelMatrix(km_train, train, ntrain, train, ntrain);
  kernelMatrix(km_test, test, ntest, train, ntrain);
  kernelMatrix(km_validate, validate, nvalidate, train, ntrain);
}

__device__ double getElement(double *m, int i, int j, int stride){
  return *(m + i * stride + j);
}

__device__ void setElement(double *m, int i, int j, int stride, double val){
  m[i * stride + j] = val;
}

__device__ int getElementInt(int *m, int i, int j, int stride){
  return *(m + i * stride + j);
}

__device__ void setElementInt(int *m, int i, int j, int stride, int val){
  m[i * stride + j] = val;
}

__device__ int getTargetByOffset(int ino, int kk){
  return target[target_offset[ino] + kk];
}

__device__ void setTargetByOffset(int ino, int kk, int t){
  target[target_offset[ino] + kk] = t;
}

__device__ int getTargetDist(int ino, int kk){
  return dist_target[target_offset[ino] + kk];
}

__device__ double calcDist(int i, double *km1, int j, double *km2){
  int tid = threadIdx.x;
  
  __shared__ double diff_k[256];
  __shared__ double sum[256];
  __shared__ double norm[64];
  
  if (tid < 64)
    norm[tid] = .0;
  
  int pos;
  for (int m = 0; m < (ntrain - 1)/blockDim.x + 1; ++ m){  
	__syncthreads();	
    pos = m * blockDim.x + tid;
	if (pos < ntrain)
      diff_k[tid] = getElement(km1, i, pos, ntrain) - getElement(km2, j, pos, ntrain);
	for (int d = 0; d < nfeat; ++ d){
	  __syncthreads();
	  if (pos < ntrain)
	    sum[tid] = getElement(O[idx_o], d, pos, ntrain) * diff_k[tid];
	  else
	    sum[tid] = .0;
		
	  int stride = blockDim.x/2;
	  while (stride > 0){
	    __syncthreads();
		if (tid < stride)
		  sum[tid] += sum[tid + stride];
		stride /= 2;
	  }
	  __syncthreads();
	  
	  if (tid == 0)
	    norm[d] += sum[0];
	}
  }
  
  if (tid < nfeat)
    norm[tid] = norm[tid]*norm[tid];
  
  __syncthreads();
  
  double s = .0;
  for (int d = 0; d < nfeat; ++ d)
	s += norm[d];
  return s;
}

__device__ void calcTargetDist(){
  int tid = threadIdx.x;
  int bid = blockIdx.x; 
  int i, j;
  if (tid == 0)
    sub_fval[bid] = .0;

  int c = 0;
  for (int m = 0; m < ntrain; ++ m)
    for (int n = 0; n < nn[label_train[m]]; ++ n){
	  i = m;
	  j = getTargetByOffset(m, n);
	  if(c%gridDim.x == bid){
	    double val = calcDist(i, km_train, j, km_train);
        if (tid == 0){
	      dist_target[target_offset[m] + n] = val;
          sub_fval[bid] += val;
	    }
	  }
	  ++ c;
	}
}

__device__ void updateDist(double *dist, struct Inst * inst1, int height, struct Inst * inst2, int width){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int i, j;
  for (int m = bid; m < height * width; m += gridDim.x){
    i = inst1[m / width].ino;
	j = inst2[m % width].ino;
	double val = calcDist(i, km_train, j, km_train);
    if (tid == 0)
	  dist[m] = val;
  }
}

__global__ void update2(){
  calcTargetDist();
  updateDist(dist1, type_inst[TN], typecount[TN], type_inst[FN], typecount[FN]);
  if (nclass == 4)
    updateDist(dist2, type_inst[TP], typecount[TP], type_inst[FP], typecount[FP]);
}

__device__ double hinge(double s){
  if (s <= -1.0)
    return .0;
  else if (s >= 0)
    return 1.0;
  else
    return 1 + s;
}

__device__ void updateTri(int idx1, int idx2, int idx3, double h){
  __syncthreads();
  for (int p = threadIdx.x; p < ntrain; p += blockDim.x)
    t_triplet[p * ntrain + idx1] += h * (getElement(km_train, idx2, p, ntrain) - getElement(km_train, idx3, p, ntrain));
}

__global__ void zeroT_triplet(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x * blockDim.x;
  for (int m = blockDim.x * bid + tid; m < ntrain * ntrain; m += size)
    t_triplet[m] = .0;  
}

__global__ void update3_2(){
  int bid = blockIdx.x;
  int i, j, l;
  double vdist, h;
  
  if (bid == 0 && threadIdx.x == 0)
    f_val = .0;
  
  for (int m = 0; m < typecount[TN] * typecount[FN]; ++ m){
    i = type_inst[TN][m / typecount[FN]].ino;
	l = type_inst[FN][m % typecount[FN]].ino;
    for (int kk = 0; kk < nn[TN]; ++ kk){
	  j = getTargetByOffset(i, kk);
	  vdist = 1 + dist_target[target_offset[i] + kk] - dist1[m];
	  if (vdist > 0 && blockIdx.x == 0 && threadIdx.x == 0)
	    f_val += vdist;
      h = hinge(vdist);
	  if (h > 0){
	    //if (label_train[i] == TP)
		h *= d_nu[label_train[i]];
	    if (i % gridDim.x == bid)
		  updateTri(i, l, j, h);
	    if (j % gridDim.x == bid)
		  updateTri(j, j, i, h);
	    if (l % gridDim.x == bid)
		  updateTri(l, i, l, h);
	  }
	}
	
    l = type_inst[TN][m / typecount[FN]].ino;
	i = type_inst[FN][m % typecount[FN]].ino;
    for (int kk = 0; kk < nn[FN]; ++ kk){
	  j = getTargetByOffset(i, kk);
	  vdist = 1 + dist_target[target_offset[i] + kk] - dist1[m];
	  if (vdist > 0 && blockIdx.x == 0 && threadIdx.x == 0)
	    f_val += vdist;
      h = hinge(vdist);
	  if (h > 0){
	    //if (label_train[i] == TP)
		h *= d_nu[label_train[i]];
	    if (i % gridDim.x == bid)
		  updateTri(i, l, j, h);
	    if (j % gridDim.x == bid)
		  updateTri(j, j, i, h);
	    if (l % gridDim.x == bid)
		  updateTri(l, i, l, h);
	  }
	}
  }
  
  if (nclass == 4){
  
  for (int m = 0; m < typecount[TP] * typecount[FP]; ++ m){
    i = type_inst[TP][m / typecount[FP]].ino;
	l = type_inst[FP][m % typecount[FP]].ino;
    for (int kk = 0; kk < nn[TP]; ++ kk){
	  j = getTargetByOffset(i, kk);
	  vdist = 1 + dist_target[target_offset[i] + kk] - dist2[m];
	  if (vdist > 0 && blockIdx.x == 0 && threadIdx.x == 0)
	    f_val += vdist;
      h = hinge(vdist);
	  if (h > 0){
		h *= d_nu[label_train[i]];
	    if (i % gridDim.x == bid)
		  updateTri(i, l, j, h);
	    if (j % gridDim.x == bid)
		  updateTri(j, j, i, h);
	    if (l % gridDim.x == bid)
		  updateTri(l, i, l, h);
	  }
	}
	
    l = type_inst[TP][m / typecount[FP]].ino;
	i = type_inst[FP][m % typecount[FP]].ino;
    for (int kk = 0; kk < nn[FP]; ++ kk){
	  j = getTargetByOffset(i, kk);
	  vdist = 1 + dist_target[target_offset[i] + kk] - dist2[m];
	  if (vdist > 0 && blockIdx.x == 0 && threadIdx.x == 0)
	    f_val += vdist;
      h = hinge(vdist);
	  if (h > 0){
		h *= d_nu[label_train[i]];
	    if (i % gridDim.x == bid)
		  updateTri(i, l, j, h);
	    if (j % gridDim.x == bid)
		  updateTri(j, j, i, h);
	    if (l % gridDim.x == bid)
		  updateTri(l, i, l, h);
	  }
	}
  }
  
  }
}

__global__ void calcFval(){ 
  if (blockIdx.x == 0 && threadIdx.x == 0)
	for (int i = 0; i < gridDim.x; ++ i)
	  f_val += sub_fval[i];
}

__global__ void updateUpdateTerm(double alpha){
  int size = gridDim.x * blockDim.x;
  for (int m = blockIdx.x * blockDim.x + threadIdx.x; m < ntrain * ntrain; m += size){
    if (m/ntrain == m%ntrain)
      t_update[m] = 1 - 2 * alpha * (t_target[m] + d_mu * t_triplet[m]);
	else
      t_update[m] = - 2 * alpha * (t_target[m] + d_mu * t_triplet[m]);
  }
}

__global__ void copyO(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x * blockDim.x;
  for (int m = blockDim.x * bid + tid; m < nfeat * ntrain; m += size)
    O[idx_o][m] = O[1 - idx_o][m];
}

__global__ void zeroO(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x * blockDim.x;
  for (int m = blockDim.x * bid + tid; m < nfeat * ntrain; m += size)
    O[1 - idx_o][m] = .0; 
}

__global__ void updateO1(){
  int tid = threadIdx.x;
  int bid_row = blockIdx.x;
  int bid_col = blockIdx.y;
  int workingtid = min(BSIZE, ntrain - bid_col * BSIZE);
  
  if (tid < workingtid)
    O[1 - idx_o][bid_row * ntrain + bid_col * BSIZE + tid] = .0;
  
  
  for (int start = 0; start < ntrain; start += BSIZE){
	int len = min(BSIZE, ntrain - start);	
    for (int i = 0; i < len; ++ i){
	if (tid < workingtid){
	  double val = getElement(O[idx_o], bid_row, start + i, ntrain) * getElement(t_update, i + start, bid_col * BSIZE + tid, ntrain);
	  O[1 - idx_o][bid_row * ntrain + bid_col * BSIZE + tid] += val;
	}
	}
  }
}

__global__ void knnUpdateDist(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x;
	
  for(int m = bid; m < ntest * ntrain; m += size){
    int i = m / ntrain;
	int j = m % ntrain;
	
	double d = DBL_MAX;
	if (nclass == 2)
	  d = calcDist(i, km_test, j, km_train);
	else{
	  if (label_test[i] == label_train[j] || label_test[i] + label_train[j] == 3)
	    d = calcDist(i, km_test, j, km_train);
	}
	
	if (tid == 0){
	  ino_knn[m] = j;
      dist_knn[m] = d;
	}
  }
}

// lauched with # block = ntest
__global__ void knnFindNeighbor(){
  
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int len = ntrain / BSIZE;
  int start = tid * len;
  if (tid < ntrain % BSIZE){
    start += tid;
	++ len;
  }
  else
    start += ntrain % BSIZE;
  
  __syncthreads();
  int b = min(len, nnegibor);
  for (int i = 0; i < b; ++ i)
    for (int j = start; j < start + len - i - 1; ++ j)
	  if(getElement(dist_knn, bid, j, ntrain) < getElement(dist_knn, bid, j + 1, ntrain)){
	    double tmp_dist = getElement(dist_knn, bid, j, ntrain);
		setElement(dist_knn, bid, j, ntrain, getElement(dist_knn, bid, j + 1, ntrain));
		setElement(dist_knn, bid, j + 1, ntrain, tmp_dist);
		
		int tmp_ino = getElementInt(ino_knn, bid, j, ntrain);
		setElementInt(ino_knn, bid, j, ntrain, getElementInt(ino_knn, bid, j + 1, ntrain));
		setElementInt(ino_knn, bid, j + 1, ntrain, tmp_ino);
	  }

  __syncthreads();  

  __shared__ double dist[BSIZE];
  __shared__ int ino[BSIZE];
  __shared__ int shortest[BSIZE];
  
  int p = start + len -1;
  for (int i = 0; i < nnegibor; ++ i){
    if (b > 0){
      dist[tid] = getElement(dist_knn, bid, p, ntrain);
      ino[tid] = getElementInt(ino_knn, bid, p, ntrain);
	}
	else
      dist[tid] = DBL_MAX;
	
    shortest[tid] = tid;
  
	int stride = blockDim.x/2;
	while (stride > 0){
	  __syncthreads();
	  if (tid < stride){
		if (dist[tid] > dist[tid + stride]){
		  dist[tid] = dist[tid + stride];
		  ino[tid] = ino[tid + stride];
		  shortest[tid] = shortest[tid + stride];
		}
	  }
	  stride /= 2;
	}
	
	__syncthreads();
	if(tid == 0)
	  setElementInt(neighbor_knn, bid, i, nnegibor, ino[0]);
	if(tid == shortest[0]){
	  -- b;
	  -- p;
	}
  }
}

__global__ void knnMatching(){
  int ub = ntest * nnegibor;
  int stride = blockDim.x * gridDim.x;  
  int idx_test, idx_train;
  for (int m = blockIdx.x * blockDim.x + threadIdx.x; m < ub; m += stride){
    idx_test = m / nnegibor;
	idx_train = neighbor_knn[m];
	if (label_test[idx_test] == label_train[idx_train])
	  neighbor_knn[m] = 1;
	else
	  neighbor_knn[m] = 0;
  }
}

// lauch with single block
__global__ void knnAcc(int neiborhood_size){
  int tid = threadIdx.x;
  int stride = blockDim.x;
  
  if (tid < 4)
    hits[tid] = 0;
	
  __shared__ int matched[BSIZE];
  matched[tid] = 0;
  
  for (int m = tid; m < ntest; m += stride){
    int nsametype = 0;
    for (int i = 0; i < neiborhood_size; ++ i)
	  nsametype += neighbor_knn[m * nnegibor + i];
	if (nsametype > neiborhood_size/2){
	  matched[tid] += 1;
	  if (label_test[m] == FN || label_test[m] == FP)
	    atomicAdd(&hits[label_test[m]], 1);
	}
	else{
	  if (label_test[m] == TN || label_test[m] == TP)
	    atomicSub(&hits[label_test[m]], 1);
	}
  }
  
  int stride1 = blockDim.x/2;
  while (stride1 > 0){
	__syncthreads();
	if (tid < stride1)
	  matched[tid] += matched[tid + stride1];
	stride1 /= 2;
  }
  
  __syncthreads();  
  if (tid ==0)
    acc_knn = 1.0 * matched[0] / ntest;
}

__global__ void knnUpdateDist_fortargetupdate(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x;
	
  for(int m = bid; m < ntrain * ntrain; m += size){
    int i = m / ntrain;
	int j = m % ntrain;
	double d = DBL_MAX;
	if (i != j && label_train[i] == label_train[j])
	  d = calcDist(i, km_train, j, km_train);
	if (tid == 0){
	  ino_knn[m] = j;
      dist_knn[m] = d;
	}
  }
}

__global__ void knnUpdateDist_train(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x;
	
  for(int m = bid; m < ntrain * ntrain; m += size){
    int i = m / ntrain;
	int j = m % ntrain;
	double d = DBL_MAX;
	if (i != j)
	  if (nclass == 2)
	    d = calcDist(i, km_train, j, km_train);
	  else
	    if (label_train[i] == label_train[j] || label_train[i] + label_train[j] == 3)
	      d = calcDist(i, km_train, j, km_train);
	if (tid == 0){
	  ino_knn[m] = j;
      dist_knn[m] = d;
	}
  }
}

// lauched with # block = ntrain
__global__ void knnFindNeighbor_train(){
  
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int len = ntrain / BSIZE;
  int start = tid * len;
  if (tid < ntrain % BSIZE){
    start += tid;
	++ len;
  }
  else
    start += ntrain % BSIZE;
  
  __syncthreads();
  int b = min(len, nnegibor);
  /* each thread sort its own chunk (start, len) by bubble sorting for b iterations.
     First b elements of ino_knn hold the closest b neighbors.*/
  for (int i = 0; i < b; ++ i)
    for (int j = start; j < start + len - i - 1; ++ j)
	  if(getElement(dist_knn, bid, j, ntrain) < getElement(dist_knn, bid, j + 1, ntrain)){
	    double tmp_dist = getElement(dist_knn, bid, j, ntrain);
		setElement(dist_knn, bid, j, ntrain, getElement(dist_knn, bid, j + 1, ntrain));
		setElement(dist_knn, bid, j + 1, ntrain, tmp_dist);
		
		int tmp_ino = getElementInt(ino_knn, bid, j, ntrain);
		setElementInt(ino_knn, bid, j, ntrain, getElementInt(ino_knn, bid, j + 1, ntrain));
		setElementInt(ino_knn, bid, j + 1, ntrain, tmp_ino);
	  }

  __syncthreads();  

  __shared__ double dist[BSIZE];
  __shared__ int ino[BSIZE];
  __shared__ int shortest[BSIZE];
  
  /* perform a merge sort of BSIZE sorted chunk. */
  int p = start + len -1;
  for (int i = 0; i < nnegibor; ++ i){
    if (b > 0){
      dist[tid] = getElement(dist_knn, bid, p, ntrain);
      ino[tid] = getElementInt(ino_knn, bid, p, ntrain);
	}
	else
      dist[tid] = DBL_MAX;
	
    shortest[tid] = tid;
  
	int stride = blockDim.x/2;
	while (stride > 0){
	  __syncthreads();
	  if (tid < stride){
		if (dist[tid] > dist[tid + stride]){
		  dist[tid] = dist[tid + stride];
		  ino[tid] = ino[tid + stride];
		  shortest[tid] = shortest[tid + stride];
		}
	  }
	  stride /= 2;
	}
	
	__syncthreads();
	if(tid == 0)
	  setElementInt(neighbor_knn, bid, i, nnegibor, ino[0]);
	if(tid == shortest[0]){
	  -- b;
	  -- p;
	}
  }
}


__global__ void knnMatching_train(){
  int ub = ntrain * nnegibor;
  int stride = blockDim.x * gridDim.x;  
  int idx_train1, idx_train2;
  for (int m = blockIdx.x * blockDim.x + threadIdx.x; m < ub; m += stride){
    idx_train1 = m / nnegibor;
	idx_train2 = neighbor_knn[m];
	if (label_train[idx_train1] == label_train[idx_train2])
	  neighbor_knn[m] = 1;
	else
	  neighbor_knn[m] = 0;
  }
}

// lauch with single block
__global__ void knnAcc_train(int neiborhood_size){
  int tid = threadIdx.x;
  int stride = blockDim.x;
  
  if (tid < 4)
    hits[tid] = 0;
	
  __shared__ int matched[BSIZE];
  matched[tid] = 0;
  
  for (int m = tid; m < ntrain; m += stride){
    int nsametype = 0;
    for (int i = 0; i < neiborhood_size; ++ i)
	  nsametype += neighbor_knn[m * nnegibor + i];
	if (nsametype > neiborhood_size/2){
	  matched[tid] += 1;
	  if (label_train[m] == FN || label_train[m] == FP)
	    atomicAdd(&hits[label_train[m]], 1);
	}
	else{
	  if (label_train[m] == TN || label_train[m] == TP)
	    atomicSub(&hits[label_train[m]], 1);
	}
  }
  
  int stride1 = blockDim.x/2;
  while (stride1 > 0){
	__syncthreads();
	if (tid < stride1)
	  matched[tid] += matched[tid + stride1];
	stride1 /= 2;
  }
  
  __syncthreads();  
  if (tid ==0)
    acc_knn = 1.0 * matched[0] / ntrain;
}

///////////////////////////////////////////////////////////////////////

__global__ void knnUpdateDist1(Type eType){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x;
  int ninst;
  double *km;
  short *label;
  switch (eType){
    case TRAIN:
	  ninst = ntrain;
	  km = km_train;
	  label = label_train;
	  break;
    case VALIDATE:
	  ninst = nvalidate;
	  km = km_validate;
	  label = label_validate;
	  break;
    case TEST:
	  ninst = ntest;
	  km = km_test;
	  label = label_test;
	  break;
  }
	
  for(int m = bid; m < ninst * ntrain; m += size){
    int i = m / ntrain;
	int j = m % ntrain;
	double d = DBL_MAX;
	if (nclass == 2){
	  if (eType != TRAIN || i != j)
	    d = calcDist(i, km, j, km_train);
	}
	else{
	  if (eType != TRAIN || i != j)
	    if (label[i] == label_train[j] || label_train[i] + label_train[j] == 3)
	      d = calcDist(i, km, j, km_train);
	}
	if (tid == 0){
	  ino_knn[m] = j;
      dist_knn[m] = d;
	}
  }
}

// lauched with # block = ntrain
__global__ void knnFindNeighbor1(){
  
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int len = ntrain / BSIZE;
  int start = tid * len;
  if (tid < ntrain % BSIZE){
    start += tid;
	++ len;
  }
  else
    start += ntrain % BSIZE;
  
  __syncthreads();
  int b = min(len, nnegibor);
  /* each thread sort its own chunk (start, len) by bubble sorting for b iterations.
     First b elements of ino_knn hold the closest b neighbors.*/
  for (int i = 0; i < b; ++ i)
    for (int j = start; j < start + len - i - 1; ++ j)
	  if(getElement(dist_knn, bid, j, ntrain) < getElement(dist_knn, bid, j + 1, ntrain)){
	    double tmp_dist = getElement(dist_knn, bid, j, ntrain);
		setElement(dist_knn, bid, j, ntrain, getElement(dist_knn, bid, j + 1, ntrain));
		setElement(dist_knn, bid, j + 1, ntrain, tmp_dist);
		
		int tmp_ino = getElementInt(ino_knn, bid, j, ntrain);
		setElementInt(ino_knn, bid, j, ntrain, getElementInt(ino_knn, bid, j + 1, ntrain));
		setElementInt(ino_knn, bid, j + 1, ntrain, tmp_ino);
	  }

  __syncthreads();  

  __shared__ double dist[BSIZE];
  __shared__ int ino[BSIZE];
  __shared__ int shortest[BSIZE];
  
  /* perform a merge sort of BSIZE sorted chunk. */
  int p = start + len -1;
  for (int i = 0; i < nnegibor; ++ i){
    if (b > 0){
      dist[tid] = getElement(dist_knn, bid, p, ntrain);
      ino[tid] = getElementInt(ino_knn, bid, p, ntrain);
	}
	else
      dist[tid] = DBL_MAX;
	
    shortest[tid] = tid;
  
	int stride = blockDim.x/2;
	while (stride > 0){
	  __syncthreads();
	  if (tid < stride){
		if (dist[tid] > dist[tid + stride]){
		  dist[tid] = dist[tid + stride];
		  ino[tid] = ino[tid + stride];
		  shortest[tid] = shortest[tid + stride];
		}
	  }
	  stride /= 2;
	}
	
	__syncthreads();
	if(tid == 0)
	  setElementInt(neighbor_knn, bid, i, nnegibor, ino[0]);
	if(tid == shortest[0]){
	  -- b;
	  -- p;
	}
  }
}


__global__ void knnMatching1(Type eType){
  int ninst;
  short *label;
  switch (eType){
    case TRAIN:
	  ninst = ntrain;
	  label = label_train;
	  break;
    case VALIDATE:
	  ninst = nvalidate;
	  label = label_validate;
	  break;
    case TEST:
	  ninst = ntest;
	  label = label_test;
	  break;
  }
	
  int ub = ninst * nnegibor;
  int stride = blockDim.x * gridDim.x;  
  int idx1, idx2;
  for (int m = blockIdx.x * blockDim.x + threadIdx.x; m < ub; m += stride){
    idx1 = m / nnegibor;
	idx2 = neighbor_knn[m];
	if (label[idx1] == label_train[idx2])
	  neighbor_knn[m] = 1;
	else
	  neighbor_knn[m] = 0;
  }
}

// lauch with single block
__global__ void knnAcc1(int neiborhood_size, Type eType){
  int ninst;
  short *label;
  switch (eType){
    case TRAIN:
	  ninst = ntrain;
	  label = label_train;
	  break;
    case VALIDATE:
	  ninst = nvalidate;
	  label = label_validate;
	  break;
    case TEST:
	  ninst = ntest;
	  label = label_test;
	  break;
  }
  
  int tid = threadIdx.x;
  int stride = blockDim.x;
  
  if (tid < 4)
    hits[tid] = 0;
	
  __shared__ int matched[BSIZE];
  matched[tid] = 0;
  
  for (int m = tid; m < ninst; m += stride){
    int nsametype = 0;
    for (int i = 0; i < neiborhood_size; ++ i)
	  nsametype += neighbor_knn[m * nnegibor + i];
	if (nsametype > neiborhood_size/2){
	  matched[tid] += 1;
	  if (label[m] == FN || label[m] == FP)
	    atomicAdd(&hits[label[m]], 1);
	}
	else{
	  if (label[m] == TN || label[m] == TP)
	    atomicSub(&hits[label[m]], 1);
	}
  }
  
  int stride1 = blockDim.x/2;
  while (stride1 > 0){
	__syncthreads();
	if (tid < stride1)
	  matched[tid] += matched[tid + stride1];
	stride1 /= 2;
  }
  
  __syncthreads();  
  if (tid ==0)
    acc_knn = 1.0 * matched[0] / ninst;
}

///////////////////////////////////////////////////////////////////////

__global__ void updateTarget(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x * blockDim.x;
  int max_nn = max(max(nn[0], nn[1]), max(nn[2], nn[3]));
  for (int m = blockDim.x * bid + tid; m < ntrain * max_nn; m += size){
    int ino = m / max_nn;
	int idx_neighbor = m % max_nn;
    if (idx_neighbor < nn[label_train[ino]])
	  setTargetByOffset(ino, idx_neighbor, getElementInt(neighbor_knn, ino, idx_neighbor, nnegibor));
  }
}

__global__ void zeroTargetTerm(){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size = gridDim.x * blockDim.x;
  for (int m = blockDim.x * bid + tid; m < ntrain * ntrain; m += size)
    t_target[m] = .0;
}

__device__ void updateTar(int idx1, int idx2, double h){
  __syncthreads();
  for (int p = threadIdx.x; p < ntrain; p += blockDim.x)
    t_target[p * ntrain + idx1] += h * (getElement(km_train, idx1, p, ntrain) - getElement(km_train, idx2, p, ntrain));
}

__global__ void updateTargetTerm(){  
  int i, j;
  double h;
  int bid = blockIdx.x;
  
  for (i = 0; i < ntrain; ++ i){
    for (int kk = 0; kk < nn[label_train[i]]; ++ kk){
	  j = getTargetByOffset(i, kk);
	  
		h = d_nu[label_train[i]];
	    if (i % gridDim.x == bid)
		  updateTar(i, j, h);
	    if (j % gridDim.x == bid)
		  updateTar(j, i, h);

	}
  }
}

__global__ void countTarget(){
  __shared__ int stay[BSIZE*4];
  
  int tid = threadIdx.x;

  for (int i = 0; i < 4; ++ i)
    stay[tid + BSIZE * i] = 0;
  
  for(int m = tid; m < ntrain; m += BSIZE){
    int l = label_train[m];
	for (int i = 0; i < nn[l]; ++ i){
	  int n = getElementInt(neighbor_knn, m, i, nnegibor);
	  for (int j = 0; j < nn[l]; ++ j){
	    int t = getTargetByOffset(m, j);
	    if ( n == t)
		  ++ stay[l * BSIZE + tid];
	  }
	}
  }
  
  for (int i = 0; i < 4; ++ i){
    int stride1 = blockDim.x/2;
    while (stride1 > 0){
	  __syncthreads();
	  if (tid < stride1)
	    stay[BSIZE * i + tid] += stay[BSIZE * i + tid + stride1];
	  stride1 /= 2;
    }
    __syncthreads();
    if (tid == 0)
	  hits[i] = stay[BSIZE * i];
  }
}

void deviceInitKernelMatrix(int nf, int trainninst, int testninst, int validateninst, double *traindata, double *testdata, double *validatedata){

  //hipMemcpyToSymbol(HIP_SYMBOL(ntrain), trainninst, sizeof(int), 0, hipMemcpyHostToDevice);
  //hipMemcpyToSymbol(HIP_SYMBOL(ntest), testninst, sizeof(int), 0, hipMemcpyHostToDevice);
  //hipMemcpyToSymbol(HIP_SYMBOL(nfeat), nf, sizeof(int), 0, hipMemcpyHostToDevice);
  
  
  double *d_train_data, *d_test_data, *d_validate_data;
  hipMalloc((void **)&d_train_data, sizeof(double) * trainninst * nf);
  hipMemcpy(d_train_data, traindata, sizeof(double) * trainninst * nf, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_test_data, sizeof(double) * testninst * nf);
  hipMemcpy(d_test_data, testdata, sizeof(double) * testninst * nf, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_validate_data, sizeof(double) * validateninst * nf);
  hipMemcpy(d_validate_data, validatedata, sizeof(double) * validateninst * nf, hipMemcpyHostToDevice);
  
  double *d_kernel_matrix_train, *d_kernel_matrix_test, *d_kernel_matrix_validate;
  hipMalloc((void **)&d_kernel_matrix_train, sizeof(double) * trainninst * trainninst);
  hipMemcpyToSymbol(HIP_SYMBOL(km_train), &d_kernel_matrix_train, sizeof(double*), 0, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_kernel_matrix_test, sizeof(double) * testninst * trainninst);
  hipMemcpyToSymbol(HIP_SYMBOL(km_test), &d_kernel_matrix_test, sizeof(double*), 0, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_kernel_matrix_validate, sizeof(double) * validateninst * trainninst);
  hipMemcpyToSymbol(HIP_SYMBOL(km_validate), &d_kernel_matrix_validate, sizeof(double*), 0, hipMemcpyHostToDevice);
  
  // Run the event recording
  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event) ;
  hipEventCreate(&stop_event) ;
  hipEventRecord(start_event, 0);
  
  calcKM<<<84, 256>>>(d_train_data, d_test_data, d_validate_data);
  hipDeviceSynchronize();
  
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  
  hipFree(d_train_data);
  hipFree(d_test_data);
  hipFree(d_validate_data);
}

void mallocToSymbol(size_t data_size, size_t offset, size_t pointer_size, const char *deviceSymbol){
  void *d_data;
  hipMalloc((void **)&d_data, data_size);
  hipMemcpyToSymbol(HIP_SYMBOL(deviceSymbol), &d_data, pointer_size, offset, hipMemcpyHostToDevice);
}

void dataToSymbol(void *data, size_t data_size, size_t offset, size_t pointer_size, const char *deviceSymbol){
  if (pointer_size > 0){
    void *d_data;
    hipMalloc((void **)&d_data, data_size);
    hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(deviceSymbol), &d_data, pointer_size, offset, hipMemcpyHostToDevice);
  }
  else
    hipMemcpyToSymbol(HIP_SYMBOL(deviceSymbol), data, data_size, offset, hipMemcpyHostToDevice);
}

void deviceInitInstList(struct Inst *inst, unsigned *count, unsigned ninst){
  
  struct Inst *gi[4];
  for (int i = 0; i < 4; ++ i){
    if (count[i] > 0)
      gi[i] = (struct Inst *)malloc(sizeof(struct Inst) * count[i]);
  }

  int p[4] = {0, 0, 0, 0};
  for(int i = 0; i < ninst; ++ i){
    int type = inst[i].label;
	gi[type][p[type]].ino = inst[i].ino;
	gi[type][p[type]].label = inst[i].label;
	++ p[type];
  }
  
  struct Inst *d_inst;
  hipMalloc((void **)&d_inst, sizeof(struct Inst) * ninst);
  unsigned start = 0;
  for (int i = 0; i < 4; ++ i){
    if (count[i] > 0)
	  hipMemcpy(d_inst + start, gi[i], sizeof(struct Inst) * count[i], hipMemcpyHostToDevice);
    struct Inst *dd_inst = d_inst + start;
    hipMemcpyToSymbol(HIP_SYMBOL(type_inst), &dd_inst, sizeof(struct Inst *), i * sizeof(struct Inst *), hipMemcpyHostToDevice);
    start += count[i];
  }
  
  for (int i = 0; i < 4; ++ i){
    if (count[i] > 0)
      free(gi[i]);
  }
}


int totalMissed;
double targetCoverage[4];
double minCoverage;
int super = 0;

unsigned iter = 0;
unsigned n_target_update = 0;
/*
double global_max_acc = .0;
unsigned global_max_iter = 0;
unsigned global_max_pos = 0;
double global_max_acc_train = .0;
unsigned global_max_iter_train = 0;
unsigned global_max_pos_train = 0;
*/

double global_max_acc1[3] = {.0};
double global_max_pos1[3] = {.0};
double global_max_iter1[3] = {.0};

int targetUpdateNeeded(){
  if (super){
    super = 0;
	return 1;
  }
  if (alpha < 1e-8 && totalMissed > 0)
  //if ((alpha < 1e-8 && totalMissed > 0) || minCoverage < 0.5)
    return 1;
  else
    return 0;
}

void findNeighbor(Type eType){
  int ninst;
  switch (eType){
    case TRAIN:
	  ninst = n_train;
	  break;
    case VALIDATE:
	  ninst = n_validate;
	  break;
    case TEST:
	  ninst = n_test;
	  break;
  }
  knnUpdateDist1<<<84, BSIZE>>>(eType);
  knnFindNeighbor1<<<ninst, BSIZE>>>();
}

void calcAcc(Type eType){
  double dd[20];
  int h_hits[4];

  knnMatching1<<<84, BSIZE>>>(eType);

  double max_acc = .0;
  int max_acc_k = -1;
  for (int i = 0; i < 20; ++ i){
    knnAcc1<<<1, BSIZE>>>(2 * i + 1, eType);
	hipDeviceSynchronize();
	hipMemcpyFromSymbol(h_hits, HIP_SYMBOL(hits), sizeof(int) * 4, 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dd[i], HIP_SYMBOL(acc_knn), sizeof(double), 0, hipMemcpyDeviceToHost);

	if (dd[i] > max_acc){
	  max_acc = dd[i];
	  max_acc_k = 2 * i + 1;
    }
    cout << h_hits[0] + h_hits[1] + h_hits[2] + h_hits[3] << "(" << h_hits[0] << "," << h_hits[1] << "," << h_hits[2] << "," << h_hits[3] << "), ";
  }
	
  //if (max_acc >= global_max_acc && iter > 10){
  if (max_acc >= global_max_acc1[eType]){
      global_max_acc1[eType] = max_acc;
	  global_max_iter1[eType] = iter;
	  global_max_pos1[eType] = max_acc_k;
  }
    cout << endl << "max acc = " << max_acc << " at k = " << max_acc_k 
    << ". global max = " << global_max_acc1[eType] << " in iter " << global_max_iter1[eType] << " at k = " << global_max_pos1[eType] << endl;
}

void findTarget(){
  int h_hits[4];
    countTarget<<<1, BSIZE>>>();
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(h_hits, HIP_SYMBOL(hits), sizeof(int) * 4, 0, hipMemcpyDeviceToHost);
    cout << "Targets: " 
	<< 1.0 * h_hits[0]/(tcount[0]*k[0]) << "(" << h_hits[0] << "/" << tcount[0]*k[0] << "), " 
	<< 1.0 * h_hits[1]/(tcount[1]*k[1]) << "(" << h_hits[1] << "/" << tcount[1]*k[1] << "), " 
	<< 1.0 * h_hits[2]/(tcount[2]*k[2]) << "(" << h_hits[2] << "/" << tcount[2]*k[2] << "), " 
	<< 1.0 * h_hits[3]/(tcount[3]*k[3]) << "(" << h_hits[3] << "/" << tcount[3]*k[3] << ")"<< endl ;	
	
	minCoverage = 1.0;
    for (int i = 0; i < 4; ++ i){
      targetCoverage[i] = 1.0 * h_hits[i] / (tcount[i]*k[i]);
	  if (minCoverage > targetCoverage[i])
	    minCoverage = targetCoverage[i];
    }

	totalMissed = 0;
    for (int i = 0; i < 4; ++ i)
      totalMissed += tcount[i] * k[i] - h_hits[i];
}
	  
void kernelTest(){
  char path[1024];
  getcwd(path, 1024);
  double original_alpha = alpha;
  double f, f_old = DBL_MAX;
  //int h_hits[4];
  //deviceInitKnn(n, n_test, 40);
    knnFindNeighbor_train<<<n_train, BSIZE>>>();
    updateTarget<<<84, BSIZE>>>();
	zeroTargetTerm<<<84, BSIZE>>>();
	updateTargetTerm<<<84, BSIZE>>>();
	
  unsigned min_iter = 0;  
  bool targetUpdated = false;
  int idx = 1;
  
  while(true){
  // Run the event recording
  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  
  cout << endl << "Iter = " << iter << ", mu = " << mu << ", k = " << k[0] << "," << k[1] << "," << k[2] << "," << k[3] << ", nu = "  << nu[0] << "," << nu[1] << "," << nu[2] << "," << nu[3] << endl;  

  idx = 1 - idx;
  hipMemcpyToSymbol(HIP_SYMBOL(idx_o), &idx, sizeof(int), 0, hipMemcpyHostToDevice);
  
  
  // update target and target term periodically
  if (targetUpdateNeeded()){
  //if (false){
    knnUpdateDist_fortargetupdate<<<84, BSIZE>>>();
    knnFindNeighbor_train<<<n_train, BSIZE>>>();
    updateTarget<<<84, BSIZE>>>();
	zeroTargetTerm<<<84, BSIZE>>>();
	updateTargetTerm<<<84, BSIZE>>>();
	alpha = original_alpha;
    targetUpdated = true;
	n_target_update ++;
  }
  
  // update distances to targets(i,j) and between opposing points(i,l)
  update2<<<84, 256>>>();
  
  // update t_triplet by calculating vdist of every (i, j, l)
  zeroT_triplet<<<84, 256>>>(); 
  update3_2<<<84, 256>>>();
  
  // update object function value
  calcFval<<<84, 256>>>();
  
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&f, HIP_SYMBOL(f_val), sizeof(double), 0, hipMemcpyDeviceToHost);
  cout << "f_val= " << f;
  
  if (f < f_old || targetUpdated){
    targetUpdated = false;
	cout << ", reduced by " << f_old - f << endl;
    f_old = f;
	min_iter = iter;
    alpha *= 1.1;
	
	// test dataset
	findNeighbor(TEST);
	calcAcc(TEST);
	
	// validate dataset
	findNeighbor(VALIDATE);
	calcAcc(VALIDATE);
	
	// train dataset
	findNeighbor(TRAIN);
	findTarget();
	/*
    countTarget<<<1, BSIZE>>>();
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(h_hits, HIP_SYMBOL(hits), sizeof(int) * 4, 0, hipMemcpyDeviceToHost);
    cout << "Targets: " 
	<< 1.0 * h_hits[0]/(tcount[0]*k[0]) << "(" << h_hits[0] << "/" << tcount[0]*k[0] << "), " 
	<< 1.0 * h_hits[1]/(tcount[1]*k[1]) << "(" << h_hits[1] << "/" << tcount[1]*k[1] << "), " 
	<< 1.0 * h_hits[2]/(tcount[2]*k[2]) << "(" << h_hits[2] << "/" << tcount[2]*k[2] << "), " 
	<< 1.0 * h_hits[3]/(tcount[3]*k[3]) << "(" << h_hits[3] << "/" << tcount[3]*k[3] << ")"<< endl ;	
	
	minCoverage = 1.0;
    for (int i = 0; i < 4; ++ i){
      targetCoverage[i] = 1.0 * h_hits[i] / (tcount[i]*k[i]);
	  if (minCoverage > targetCoverage[i])
	    minCoverage = targetCoverage[i];
    }

	totalMissed = 0;
    for (int i = 0; i < 4; ++ i)
      totalMissed += tcount[i] * k[i] - h_hits[i];
    */
	calcAcc(TRAIN);	
  }
  else{
	cout << ", increased by " << f - f_old;
    alpha /= 10;
    copyO<<<84, BSIZE>>>();
    update2<<<84, 256>>>();
    zeroT_triplet<<<84, 256>>>();
    update3_2<<<84, 256>>>();
  }
  
  cout << "min_f = " << f_old << " at iter " << min_iter << ", alpha = " << alpha << " target udpated = " << n_target_update << endl;

  // t_update = I - 2 * alpha * (t_target + t_triplet)
  updateUpdateTerm<<<84, 256>>>(alpha);  
  
  // update omega = omega * t_update
  zeroO<<<84, 256>>>();
  dim3 dimGrid(n_feat, (n_train - 1) / BSIZE + 1);
  dim3 dimBlock(BSIZE);
  updateO1<<<dimGrid, dimBlock>>>();  
  hipDeviceSynchronize();
  
  float time_kernel;
  hipEventRecord(stop_event, 0);
  hipEventElapsedTime(&time_kernel, start_event, stop_event);
  cout << "time " << time_kernel/1000 << " at " << path << endl;
  ++ iter;
  //if (iter > 100)
  if (alpha < 1e-10)
    break;
  }
}
